#include "caffe/layers/dense_block_layer.hpp"
#include "caffe/layers/batch_norm_layer.hpp"
#include "caffe/util/device_alternate.hpp"

namespace caffe {

//template <typename Dtype>
//void caffe_cublas_mul(const int N, const Dtype* a, const Dtype* b, Dtype* y);
//
//template <> 
//void caffe_cublas_mul<float>(const int N, const float* a, const float* b, float* y)
//{
//  float one(1.);
//  float zero(0.);
//  CUBLAS_CHECK(hipblasSsbmv(Caffe::cublas_handle(),
//    HIPBLAS_FILL_MODE_LOWER, N, 0,
//    &one, a, 1, b, 1,
//    &zero, y, 1));
//}
//template <>
//void caffe_cublas_mul<double>(const int N, const double* a, const double* b, double* y)
//{
//  double one(1.);
//  double zero(0.);
//  CUBLAS_CHECK(hipblasDsbmv(Caffe::cublas_handle(),
//    HIPBLAS_FILL_MODE_LOWER, N, 0,
//    &one, a, 1, b, 1,
//    &zero, y, 1));
//}

#ifdef USE_CUDNN
template <typename Dtype>
dense_block::StaticVariable<Dtype> dense_block::StaticVariable<Dtype>::instance_;

template <typename Dtype>
dense_block::StaticVariable<Dtype>::StaticVariable() : fast_scale_fwd_op_desc_(NULL)
{
  CUDNN_CHECK(hipdnnCreateOpTensorDescriptor(&fast_scale_fwd_op_desc_));
  CUDNN_CHECK(hipdnnSetOpTensorDescriptor(fast_scale_fwd_op_desc_, HIPDNN_OP_TENSOR_MUL, cudnn::dataType<Dtype>::type, HIPDNN_PROPAGATE_NAN));
}

template <typename Dtype>
dense_block::StaticVariable<Dtype>::~StaticVariable()
{
  if (fast_scale_fwd_op_desc_)
    CUDNN_CHECK(hipdnnDestroyOpTensorDescriptor(fast_scale_fwd_op_desc_));
}

template <typename Dtype>
static void reduce_nhw(hipdnnHandle_t handle, 
                       Dtype alpha_, hipdnnTensorDescriptor_t nchw_desc_, const Dtype* nchw_ptr_,
                       Dtype beta_, hipdnnTensorDescriptor_t c_desc_, Dtype* c_ptr_)
{
  Dtype a(alpha_);
  Dtype b(beta_);
  // we assume that these 2 tensor descriptors are configured to right data type
  CUDNN_CHECK(hipdnnConvolutionBackwardBias(handle,
    &a, nchw_desc_, nchw_ptr_, &b, c_desc_, c_ptr_));
}

template <typename Dtype>
void dense_block::ScaleLayerFastForward(hipdnnHandle_t handle,
  hipdnnTensorDescriptor_t bottom_desc, Blob<Dtype>* bottom,
  hipdnnTensorDescriptor_t top_desc, Blob<Dtype> *top,
  hipdnnTensorDescriptor_t scale_bias_desc, ScaleLayer<Dtype> *scale_layer)
{
  CHECK_NE(bottom, top) << "ScaleLayerFastForward dose not support in-place computation";

  Dtype one(1.);
  Dtype zero(0.);

  CUDNN_CHECK(hipdnnOpTensor(handle, StaticVariable<Dtype>::get().fast_scale_fwd_op_desc(),
    &one, bottom_desc, bottom->gpu_data(),
    &one, scale_bias_desc, scale_layer->blobs()[0]->gpu_data(),
    &zero, top_desc, top->mutable_gpu_data()));

  CUDNN_CHECK(hipdnnAddTensor(handle,
    &one, scale_bias_desc, scale_layer->blobs()[1]->gpu_data(),
    &one, top_desc, top->mutable_gpu_data()));

}

template <typename Dtype>
void dense_block::ScaleLayerFastBackward(hipdnnHandle_t handle,
  hipdnnTensorDescriptor_t scale_bias_desc, ScaleLayer<Dtype> *scale_layer,
  hipdnnTensorDescriptor_t top_desc, Blob<Dtype> *top,
  hipdnnTensorDescriptor_t bottom_desc, Blob<Dtype>* bottom)
{
  CHECK_NE(bottom, top) << "ScaleLayerFastForward dose not support in-place computation";
  
  Dtype one(1.);
  Dtype zero(0.);

  // gradient w.r.t bias
  reduce_nhw(handle, one, top_desc,   top->gpu_diff(),     one, scale_bias_desc, scale_layer->blobs()[1]->mutable_gpu_diff());

  // gradient w.r.t scale
  caffe_gpu_mul(bottom->count(), bottom->gpu_data(), top->gpu_diff(), bottom->mutable_gpu_diff());
  reduce_nhw(handle, one, bottom_desc, bottom->gpu_diff(), one, scale_bias_desc, scale_layer->blobs()[0]->mutable_gpu_diff());

  // gradient w.r.t bottom
  CUDNN_CHECK(hipdnnOpTensor(handle, StaticVariable<Dtype>::get().fast_scale_fwd_op_desc(),
    &one, top_desc, top->gpu_diff(),
    &one, scale_bias_desc, scale_layer->blobs()[0]->gpu_data(),
    &zero, bottom_desc, bottom->mutable_gpu_diff()));
}

namespace dense_block {
  template void ScaleLayerFastForward(hipdnnHandle_t handle,
    hipdnnTensorDescriptor_t bottom_desc, Blob<float>* bottom,
    hipdnnTensorDescriptor_t top_desc, Blob<float> *top,
    hipdnnTensorDescriptor_t scale_bias_desc, ScaleLayer<float> *scale_layer);
  template void ScaleLayerFastForward(hipdnnHandle_t handle,
    hipdnnTensorDescriptor_t bottom_desc, Blob<double>* bottom,
    hipdnnTensorDescriptor_t top_desc, Blob<double> *top,
    hipdnnTensorDescriptor_t scale_bias_desc, ScaleLayer<double> *scale_layer);
  template void ScaleLayerFastBackward(hipdnnHandle_t handle,
    hipdnnTensorDescriptor_t scale_bias_desc, ScaleLayer<float> *scale_layer,
    hipdnnTensorDescriptor_t top_desc, Blob<float> *top,
    hipdnnTensorDescriptor_t bottom_desc, Blob<float>* bottom);
  template void ScaleLayerFastBackward(hipdnnHandle_t handle,
    hipdnnTensorDescriptor_t scale_bias_desc, ScaleLayer<double> *scale_layer,
    hipdnnTensorDescriptor_t top_desc, Blob<double> *top,
    hipdnnTensorDescriptor_t bottom_desc, Blob<double>* bottom);
} // namespace dense_block
#endif // USE_CUNN

template <typename Dtype>
inline static void caffe_gpu_copy_async(const int N, const Dtype* X, Dtype* Y, const hipStream_t& stream) {
  if (X != Y && Caffe::mode() == Caffe::GPU) {
#ifndef CPU_ONLY
      // NOLINT_NEXT_LINE(caffe/alt_fn)
      CUDA_CHECK(hipMemcpyAsync(Y, X, sizeof(Dtype) * N, hipMemcpyDeviceToDevice, stream));
#else
      NO_GPU;
#endif
  }
}

template <typename Dtype>
static void assemble_maps_gpu(const int n, const int h, const int w, const int c0, const int c_add,
                              Dtype* dst, const Dtype* new_map, const hipStream_t& stream)
{
  // c0 = #feature-maps BEFORE assemble
  // c_add = #feature-maps to be added
  const int c1 = c0 + c_add;
  const int c_stride = h * w;
  const int src_stride = c0 * c_stride;
  const int dst_stride = c1 * c_stride;
  const int new_stride = c_add * c_stride;
  
  const Dtype* new_map_ptr = new_map + (n - 1) * new_stride;
  const Dtype *src_ptr = dst + (n - 1) * src_stride;
  Dtype *dst_ptr = dst + (n - 1) * dst_stride;
  Dtype *dst_ptr_for_new = dst_ptr + src_stride;
  
  const int src_count = c0 * c_stride;
  const int new_count = c_add * c_stride;
  
  for (int i = n - 1; i >= 0; i --, 
    new_map_ptr -= new_stride, 
    src_ptr     -= src_stride,
    dst_ptr     -= dst_stride,
    dst_ptr_for_new -= dst_stride)
  {
    if (dst_ptr > src_ptr && dst_ptr - src_ptr < src_count)
      // dst_ptr is pointing within the src region [src_ptr, src_ptr + src_count]
      // directly memcpy will cause data lossing, so we copy channel by channel from back to front
    {
      const int batch = dst_ptr - src_ptr;
      int remains = src_count;
      Dtype* p_dst = dst_ptr + src_count - batch;
      const Dtype* p_src = src_ptr + src_count - batch;
      for (; remains >= batch; remains -= batch, p_dst -= batch, p_src -= batch)
        caffe_gpu_copy_async(batch, p_src, p_dst, stream);
      if (remains)
        caffe_gpu_copy_async(remains, src_ptr, dst_ptr, stream);
    }
    else
      caffe_gpu_copy_async(src_count, src_ptr, dst_ptr, stream);
    
    caffe_gpu_copy_async(new_count, new_map_ptr, dst_ptr_for_new, stream);
  }
  
}

template <typename Dtype>
static void disassemble_maps_gpu(const int n, const int h, const int w, const int c0, const int c_add,
                                 Dtype* src, Dtype* out_map, const hipStream_t& stream)
{
  // c0 = #feature-maps AFTER disassemble
  // c_add = #feature-maps in out_map
  const int c1 = c0 + c_add;
  const int c_stride = h * w;
  const int src_stride = c1 * c_stride;
  const int dst_stride = c0 * c_stride;
  const int out_stride = c_add * c_stride;
  
  Dtype* out_map_ptr = out_map;
  Dtype *dst_ptr = src;
  const Dtype *src_ptr = src;
  const Dtype *src_ptr_for_out = src_ptr + dst_stride;
  
  const int dst_count = c0 * c_stride;
  const int out_count = c_add * c_stride;
  
  for (int i = 0; i < n; i ++,
    out_map_ptr += out_stride,
    dst_ptr     += dst_stride,
    src_ptr     += src_stride,
    src_ptr_for_out += src_stride)
  {
    caffe_gpu_copy_async(out_count, src_ptr_for_out, out_map_ptr, stream);
    // CAUTION: in memcpy, there shoud not be any overlap between
    //          src memory region and dst memory region
    if (src_ptr > dst_ptr && src_ptr < dst_ptr + dst_count)
    {
      const int batch = src_ptr - dst_ptr;
      int remains = dst_count;
      Dtype* p_dst = dst_ptr;
      const Dtype* p_src = src_ptr;
      for (; remains >= batch; remains -= batch, p_dst += batch, p_src += batch)
        caffe_gpu_copy_async(batch, p_src, p_dst, stream);
      if (remains)
        caffe_gpu_copy_async(remains, p_src, p_dst, stream);
    }
    else
      caffe_gpu_copy_async(dst_count, src_ptr, dst_ptr, stream);
  }
}

template <typename Dtype>
static void assemble_maps_gpu_adding_part(const int n, const int h, const int w, const int c0, const int c_add,
                                          Dtype* dst, const Dtype* new_map, const hipStream_t& stream)
{
  // c0 = #feature-maps BEFORE assemble
  // c_add = #feature-maps to be added
  const int c1 = c0 + c_add;
  const int c_stride = h * w;
  const int src_stride = c0 * c_stride;
  const int dst_stride = c1 * c_stride;
  const int new_stride = c_add * c_stride;
  
  const Dtype* new_map_ptr = new_map + (n - 1) * new_stride;
  const Dtype *src_ptr = dst + (n - 1) * src_stride;
  Dtype *dst_ptr = dst + (n - 1) * dst_stride;
  Dtype *dst_ptr_for_new = dst_ptr + src_stride;
  
  const int src_count = c0 * c_stride;
  const int new_count = c_add * c_stride;
  
  for (int i = n - 1; i >= 0; i --, 
    new_map_ptr -= new_stride, 
    src_ptr     -= src_stride,
    dst_ptr     -= dst_stride,
    dst_ptr_for_new -= dst_stride)
  {
    //if (dst_ptr > src_ptr && dst_ptr - src_ptr < src_count)
    //  // dst_ptr is pointing within the src region [src_ptr, src_ptr + src_count]
    //  // directly memcpy will cause data lossing, so we copy channel by channel from back to front
    //{
    //  const int batch = dst_ptr - src_ptr;
    //  int remains = src_count;
    //  Dtype* p_dst = dst_ptr + src_count - batch;
    //  const Dtype* p_src = src_ptr + src_count - batch;
    //  for (; remains >= batch; remains -= batch, p_dst -= batch, p_src -= batch)
    //    caffe_gpu_copy_async(batch, p_src, p_dst, stream);
    //  if (remains)
    //    caffe_gpu_copy_async(remains, src_ptr, dst_ptr, stream);
    //}
    //else
    //  caffe_gpu_copy_async(src_count, src_ptr, dst_ptr, stream);
    
    caffe_gpu_copy_async(new_count, new_map_ptr, dst_ptr_for_new, stream);
  }
  
}

template <typename Dtype>
static void disassemble_maps_gpu_adding_part(const int n, const int h, const int w, const int c0, const int c_add,
                                             Dtype* src, Dtype* out_map, const hipStream_t& stream)
{
  // c0 = #feature-maps AFTER disassemble
  // c_add = #feature-maps in out_map
  const int c1 = c0 + c_add;
  const int c_stride = h * w;
  const int src_stride = c1 * c_stride;
  const int dst_stride = c0 * c_stride;
  const int out_stride = c_add * c_stride;
  
  Dtype* out_map_ptr = out_map;
  Dtype *dst_ptr = src;
  const Dtype *src_ptr = src;
  const Dtype *src_ptr_for_out = src_ptr + dst_stride;
  
  const int dst_count = c0 * c_stride;
  const int out_count = c_add * c_stride;
  
  for (int i = 0; i < n; i ++,
    out_map_ptr += out_stride,
    dst_ptr     += dst_stride,
    src_ptr     += src_stride,
    src_ptr_for_out += src_stride)
  {
    caffe_gpu_copy_async(out_count, src_ptr_for_out, out_map_ptr, stream);
    // CAUTION: in memcpy, there shoud not be any overlap between
    //          src memory region and dst memory region
    //if (src_ptr > dst_ptr && src_ptr < dst_ptr + dst_count)
    //{
    //  const int batch = src_ptr - dst_ptr;
    //  int remains = dst_count;
    //  Dtype* p_dst = dst_ptr;
    //  const Dtype* p_src = src_ptr;
    //  for (; remains >= batch; remains -= batch, p_dst += batch, p_src += batch)
    //    caffe_gpu_copy_async(batch, p_src, p_dst, stream);
    //  if (remains)
    //    caffe_gpu_copy_async(remains, p_src, p_dst, stream);
    //}
    //else
    //  caffe_gpu_copy_async(dst_count, src_ptr, dst_ptr, stream);
  }
}

template <typename Dtype>
static void assemble_maps_gpu_origin_part(const int n, const int h, const int w, const int c0, const int c_add,
                                          Dtype* dst, const Dtype* new_map, const hipStream_t& stream)
{
  // c0 = #feature-maps BEFORE assemble
  // c_add = #feature-maps to be added
  const int c1 = c0 + c_add;
  const int c_stride = h * w;
  const int src_stride = c0 * c_stride;
  const int dst_stride = c1 * c_stride;
  const int new_stride = c_add * c_stride;
  
  const Dtype* new_map_ptr = new_map + (n - 1) * new_stride;
  const Dtype *src_ptr = dst + (n - 1) * src_stride;
  Dtype *dst_ptr = dst + (n - 1) * dst_stride;
  Dtype *dst_ptr_for_new = dst_ptr + src_stride;
  
  const int src_count = c0 * c_stride;
  const int new_count = c_add * c_stride;
  
  for (int i = n - 1; i >= 0; i --, 
    new_map_ptr -= new_stride, 
    src_ptr     -= src_stride,
    dst_ptr     -= dst_stride,
    dst_ptr_for_new -= dst_stride)
  {
    if (dst_ptr > src_ptr && dst_ptr - src_ptr < src_count)
      // dst_ptr is pointing within the src region [src_ptr, src_ptr + src_count]
      // directly memcpy will cause data lossing, so we copy channel by channel from back to front
    {
      const int batch = dst_ptr - src_ptr;
      int remains = src_count;
      Dtype* p_dst = dst_ptr + src_count - batch;
      const Dtype* p_src = src_ptr + src_count - batch;
      for (; remains >= batch; remains -= batch, p_dst -= batch, p_src -= batch)
        caffe_gpu_copy_async(batch, p_src, p_dst, stream);
      if (remains)
        caffe_gpu_copy_async(remains, src_ptr, dst_ptr, stream);
    }
    else
      caffe_gpu_copy_async(src_count, src_ptr, dst_ptr, stream);
    
    //caffe_gpu_copy_async(new_count, new_map_ptr, dst_ptr_for_new, stream);
  }
  
}

template <typename Dtype>
static void disassemble_maps_gpu_origin_part(const int n, const int h, const int w, const int c0, const int c_add,
                                             Dtype* src, Dtype* out_map, const hipStream_t& stream)
{
  // c0 = #feature-maps AFTER disassemble
  // c_add = #feature-maps in out_map
  const int c1 = c0 + c_add;
  const int c_stride = h * w;
  const int src_stride = c1 * c_stride;
  const int dst_stride = c0 * c_stride;
  const int out_stride = c_add * c_stride;
  
  Dtype* out_map_ptr = out_map;
  Dtype *dst_ptr = src;
  const Dtype *src_ptr = src;
  const Dtype *src_ptr_for_out = src_ptr + dst_stride;
  
  const int dst_count = c0 * c_stride;
  const int out_count = c_add * c_stride;
  
  for (int i = 0; i < n; i ++,
    out_map_ptr += out_stride,
    dst_ptr     += dst_stride,
    src_ptr     += src_stride,
    src_ptr_for_out += src_stride)
  {
    //caffe_gpu_copy_async(out_count, src_ptr_for_out, out_map_ptr, stream);
    // CAUTION: in memcpy, there shoud not be any overlap between
    //          src memory region and dst memory region
    if (src_ptr > dst_ptr && src_ptr < dst_ptr + dst_count)
    {
      const int batch = src_ptr - dst_ptr;
      int remains = dst_count;
      Dtype* p_dst = dst_ptr;
      const Dtype* p_src = src_ptr;
      for (; remains >= batch; remains -= batch, p_dst += batch, p_src += batch)
        caffe_gpu_copy_async(batch, p_src, p_dst, stream);
      if (remains)
        caffe_gpu_copy_async(remains, p_src, p_dst, stream);
    }
    else
      caffe_gpu_copy_async(dst_count, src_ptr, dst_ptr, stream);
  }
}

template <typename Dtype>
static void updateMovingAverage(BatchNormLayer<Dtype>* layer)
{
  // this is the implementation of update moving average from caffe's BatchNorm
  Dtype moving_average_fraction_ = 
    layer->layer_param().batch_norm_param().moving_average_fraction();

  layer->blobs()[2]->mutable_cpu_data()[0] *= moving_average_fraction_;
  layer->blobs()[2]->mutable_cpu_data()[0] -= 1;
}
#if 0 // debug utils
static hipdnnTensorDescriptor_t copyTensor4dDesc(hipdnnTensorDescriptor_t tensorDesc)
{
  hipdnnTensorDescriptor_t ret;
  CUDNN_CHECK(hipdnnCreateTensorDescriptor(&ret));
  
  int n, c, h, w, ns, cs, hs, ws;
  hipdnnDataType_t type;
  CUDNN_CHECK(hipdnnGetTensor4dDescriptor(tensorDesc, &type, &n, &c, &h, &w, &ns, &cs, &hs, &ws));

  CUDNN_CHECK(hipdnnSetTensor4dDescriptor(ret, HIPDNN_TENSOR_NCHW, type, n, c, h, w));
  return ret;
}
static void dumpTensorShape(hipdnnTensorDescriptor_t tensorDesc)
{
  char tmp[20];
  string ret = "";
  int maxDim = 100, actualDim;
  vector<int> dim(maxDim, 0), stride(maxDim, 0);
  hipdnnDataType_t type;
  CUDNN_CHECK(hipdnnGetTensorNdDescriptor(tensorDesc, maxDim, &type, &actualDim, dim.data(), stride.data()));
  for (int i = 0; i < actualDim; i ++)
  {
    sprintf(tmp, "%d(%d) ", dim[i], stride[i]);
    ret += tmp;
  }
  LOG(INFO) << ret;
}
template <typename Dtype>
static void fillArray(int count, Dtype *dst, Dtype val)
{
  for (int i = 0; i < count; i ++)
    dst[i] = val;
}
template <typename Dtype>
static void fillArray(int count, Dtype *dst)
{
  for (int i = 0; i < count; i ++)
    dst[i] = (rand() % 2 ? -1 : 1) * Dtype(rand() - RAND_MAX / 2) * 20 / RAND_MAX;
}
#endif // debug utils

#ifdef USE_CUDNN

template <typename Dtype>
inline static double cudnnGetBNEps(Dtype val)
{
  double ret(val);
  return ret < HIPDNN_BN_MIN_EPSILON ? HIPDNN_BN_MIN_EPSILON : ret;
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::ForwardInference_gpu(const vector<Blob<Dtype>*>& bottom,
  const vector<Blob<Dtype>*>& top)
{
  const vector<int> shape(bottom[0]->shape());
  const int n = shape[0];
  const int k0= shape[1];
  const int h = shape[2];
  const int w = shape[3];

  // copy the input data into working space 
  caffe_copy(bottom[0]->count(), bottom[0]->gpu_data(), maps_diff_.mutable_gpu_data());

  if (pre_bn_layer_->blobs()[2]->cpu_data()[0] > 0)
    convertBatchNormParams();

  Dtype one(1.f), zero(0.f);

  for (int l = 0; l < num_layers_; l++)
  {
    vector<Blob<Dtype>*> the_input_lth(1, input_lth_[l].get());
    vector<Blob<Dtype>*> the_conv3x3_inter_l(1, conv3x3_inter_[l].get());
    vector<Blob<Dtype>*> the_output_lth(1, output_lth_[l].get());

    if (use_bottleneck_)
    {
      vector<Blob<Dtype>*> the_bottleneck_inter_l(1, bottleneck_inter_[l].get());

      {
        //updateMovingAverage( (BatchNormLayer<Dtype>*)(bottle_bn_layers_[l].get()) );
        //CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
        //  cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero,
        //  input_desc_[l], the_input_lth[0]->gpu_data(),
        //  input_desc_[l], the_conv3x3_inter_l[0]->mutable_gpu_data(),
        //  input_scale_bias_desc_[l], bottle_scale_layers_[l]->blobs()[0]->gpu_data(), bottle_scale_layers_[l]->blobs()[1]->gpu_data(),
        //  1 / bottle_bn_layers_[l]->blobs()[2]->cpu_data()[0],
        //  bottle_bn_layers_[l]->blobs()[0]->mutable_gpu_data(),
        //  bottle_bn_layers_[l]->blobs()[1]->mutable_gpu_data(),
        //  cudnnGetBNEps(bottle_bn_layers_[l]->layer_param().batch_norm_param().eps()),
        //  bottleneck_bn_mean_var_[l]->mutable_gpu_data(), bottleneck_bn_mean_var_[l]->mutable_gpu_diff()
        //));
        CUDNN_CHECK(hipdnnBatchNormalizationForwardInference(
          cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero,
          input_desc_[l], the_input_lth[0]->gpu_data(),
          input_desc_[l], the_conv3x3_inter_l[0]->mutable_gpu_data(),
          input_scale_bias_desc_[l], 
          bottle_scale_layers_[l]->blobs()[0]->gpu_data(), 
          bottle_scale_layers_[l]->blobs()[1]->gpu_data(),
          bottle_bn_layers_[l]->blobs()[0]->gpu_data(),
          bottle_bn_layers_[l]->blobs()[1]->gpu_data(),
          cudnnGetBNEps(bottle_bn_layers_[l]->layer_param().batch_norm_param().eps())
        ));
      }
      //bottle_bn_layers_[l]->Forward(the_input_lth, the_conv3x3_inter_l);
      // (in gpu) async "assemble" (original part) can start here to prepare for next conv block
      assemble_maps_gpu_origin_part(n, h, w, k0 + l * growth_rate_, growth_rate_,
        maps_diff_.mutable_gpu_data(), (const Dtype*)NULL /*output_lth_[l]->gpu_data()*/, dataCopyStream_);

      //bottle_scale_layers_[l]->Forward(the_conv3x3_inter_l, the_conv3x3_inter_l);
      bottle_relu_layers_[l]->Forward(the_conv3x3_inter_l, the_conv3x3_inter_l);

      conv1x1_layers_[l]->Forward(the_conv3x3_inter_l, the_bottleneck_inter_l);

      //bn_layers_[l]->Forward(the_bottleneck_inter_l, the_bottleneck_inter_l);
      //scale_layers_[l]->Forward(the_bottleneck_inter_l, the_bottleneck_inter_l);
      {
        caffe_copy(the_bottleneck_inter_l[0]->count(),
                   the_bottleneck_inter_l[0]->gpu_data(), 
                   bottleneck_scale_tmp_[l]->mutable_gpu_data());
        //updateMovingAverage( (BatchNormLayer<Dtype>*)(bn_layers_[l].get()) );
        //CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
        //  cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero,
        //  bottleneck_inter_desc_, bottleneck_scale_tmp_[l]->gpu_data(),
        //  bottleneck_inter_desc_, the_bottleneck_inter_l[0]->mutable_gpu_data(),
        //  bottleneck_scale_bias_desc_, scale_layers_[l]->blobs()[0]->gpu_data(), scale_layers_[l]->blobs()[1]->gpu_data(),
        //  1 / bn_layers_[l]->blobs()[2]->cpu_data()[0],
        //  bn_layers_[l]->blobs()[0]->mutable_gpu_data(),
        //  bn_layers_[l]->blobs()[1]->mutable_gpu_data(),
        //  cudnnGetBNEps(bn_layers_[l]->layer_param().batch_norm_param().eps()),
        //  bn_mean_var_[l]->mutable_gpu_data(), bn_mean_var_[l]->mutable_gpu_diff()
        //));
        CUDNN_CHECK(hipdnnBatchNormalizationForwardInference(
          cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero,
          bottleneck_inter_desc_, bottleneck_scale_tmp_[l]->gpu_data(),
          bottleneck_inter_desc_, the_bottleneck_inter_l[0]->mutable_gpu_data(),
          bottleneck_scale_bias_desc_,
          scale_layers_[l]->blobs()[0]->gpu_data(),
          scale_layers_[l]->blobs()[1]->gpu_data(),
          bn_layers_[l]->blobs()[0]->gpu_data(),
          bn_layers_[l]->blobs()[1]->gpu_data(),
          cudnnGetBNEps(bn_layers_[l]->layer_param().batch_norm_param().eps())
        ));
      }
      relu_layers_[l]->Forward(the_bottleneck_inter_l, the_bottleneck_inter_l);

      conv3x3_layers_[l]->Forward(the_bottleneck_inter_l, the_output_lth);

    }
    else
    {
      {
        //updateMovingAverage( (BatchNormLayer<Dtype>*)(bn_layers_[l].get()) );
        //CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
        //  cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero,
        //  input_desc_[l], the_input_lth[0]->gpu_data(),
        //  input_desc_[l], the_conv3x3_inter_l[0]->mutable_gpu_data(),
        //  input_scale_bias_desc_[l], scale_layers_[l]->blobs()[0]->gpu_data(), scale_layers_[l]->blobs()[1]->gpu_data(),
        //  1 / bn_layers_[l]->blobs()[2]->cpu_data()[0],
        //  bn_layers_[l]->blobs()[0]->mutable_gpu_data(),
        //  bn_layers_[l]->blobs()[1]->mutable_gpu_data(),
        //  cudnnGetBNEps(bn_layers_[l]->layer_param().batch_norm_param().eps()),
        //  bn_mean_var_[l]->mutable_gpu_data(), bn_mean_var_[l]->mutable_gpu_diff()
        //));
        CUDNN_CHECK(hipdnnBatchNormalizationForwardInference(
          cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero,
          input_desc_[l], the_input_lth[0]->gpu_data(),
          input_desc_[l], the_conv3x3_inter_l[0]->mutable_gpu_data(),
          input_scale_bias_desc_[l],
          scale_layers_[l]->blobs()[0]->gpu_data(),
          scale_layers_[l]->blobs()[1]->gpu_data(),
          bn_layers_[l]->blobs()[0]->gpu_data(),
          bn_layers_[l]->blobs()[1]->gpu_data(),
          cudnnGetBNEps(bn_layers_[l]->layer_param().batch_norm_param().eps())
        ));
      }
      //bn_layers_[l]->Forward(the_input_lth, the_conv3x3_inter_l);

      // (in gpu) async "assemble" (original part) can start here to prepare for next conv block
      assemble_maps_gpu_origin_part(n, h, w, k0 + l * growth_rate_, growth_rate_,
        maps_diff_.mutable_gpu_data(), (const Dtype*)NULL /*output_lth_[l]->gpu_data()*/, dataCopyStream_);

      //scale_layers_[l]->Forward(the_conv3x3_inter_l, the_conv3x3_inter_l);
      relu_layers_[l]->Forward(the_conv3x3_inter_l, the_conv3x3_inter_l);

      conv3x3_layers_[l]->Forward(the_conv3x3_inter_l, the_output_lth);

    }

    if (use_dropout_)
    {
      dropout_layers_[l]->Forward(the_output_lth, the_output_lth);
    }


    // (in gpu) start async "assemble" (adding part) for this conv block
    //assemble_maps(n, h, w, k0 + l * growth_rate_, growth_rate_,
    //  maps_diff_.mutable_cpu_data(), output_lth_[l]->cpu_data());
    assemble_maps_gpu_adding_part(n, h, w, k0 + l * growth_rate_, growth_rate_,
      maps_diff_.mutable_gpu_data(), output_lth_[l]->gpu_data(), dataCopyStream_);

    // (in gpu) synchronize "assemble" here so we can start next conv block
    CUDA_CHECK(hipStreamSynchronize(dataCopyStream_));
  }

  //pre_bn_layer_->Forward(vector<Blob<Dtype>*>(1, &maps_diff_), vector<Blob<Dtype>*>(1, &maps_diff_));
  //post_scale_layer_->Forward(vector<Blob<Dtype>*>(1, &maps_diff_), top);
  {
    //updateMovingAverage( (BatchNormLayer<Dtype>*)(pre_bn_layer_.get()) );
    //CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
    //  cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero,
    //  final_output_desc_, maps_diff_.gpu_data(),
    //  final_output_desc_, top[0]->mutable_gpu_data(),
    //  scale_bias_desc_, post_scale_layer_->blobs()[0]->gpu_data(), post_scale_layer_->blobs()[1]->gpu_data(),
    //  1 / pre_bn_layer_->blobs()[2]->cpu_data()[0],
    //  pre_bn_layer_->blobs()[0]->mutable_gpu_data(),
    //  pre_bn_layer_->blobs()[1]->mutable_gpu_data(),
    //  cudnnGetBNEps(pre_bn_layer_->layer_param().batch_norm_param().eps()),
    //  pre_bn_mean_var_.mutable_gpu_data(), pre_bn_mean_var_.mutable_gpu_diff()
    //));
    CUDNN_CHECK(hipdnnBatchNormalizationForwardInference(
      cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero,
      final_output_desc_, maps_diff_.gpu_data(),
      final_output_desc_, top[0]->mutable_gpu_data(),
      scale_bias_desc_, 
      post_scale_layer_->blobs()[0]->gpu_data(), 
      post_scale_layer_->blobs()[1]->gpu_data(),
      pre_bn_layer_->blobs()[0]->gpu_data(),
      pre_bn_layer_->blobs()[1]->gpu_data(),
      cudnnGetBNEps(pre_bn_layer_->layer_param().batch_norm_param().eps())
    ));
  }
  post_relu_layer_->Forward(top, top);
}

template
void DenseBlockLayer<float>::ForwardInference_gpu(const vector<Blob<float>*>& bottom,
  const vector<Blob<float>*>& top);
template
void DenseBlockLayer<double>::ForwardInference_gpu(const vector<Blob<double>*>& bottom,
  const vector<Blob<double>*>& top);

template <typename Dtype>
void DenseBlockLayer<Dtype>::convertBatchNormParams()
{
  CHECK(0 < pre_bn_layer_->blobs()[2]->cpu_data()[0]) << "already in cudnn defintion";
  caffe_gpu_scal(pre_bn_layer_->blobs()[0]->count(), 
                 1 / pre_bn_layer_->blobs()[2]->cpu_data()[0], 
                 pre_bn_layer_->blobs()[0]->mutable_gpu_data());
  caffe_gpu_scal(pre_bn_layer_->blobs()[1]->count(), 
                 1 / pre_bn_layer_->blobs()[2]->cpu_data()[0], 
                 pre_bn_layer_->blobs()[1]->mutable_gpu_data());
  pre_bn_layer_->blobs()[2]->mutable_cpu_data()[0] = -pre_bn_layer_->blobs()[2]->cpu_data()[0];

  for (size_t i = 0; i < bn_layers_.size(); i++)
  {
    CHECK(0 < bn_layers_[i]->blobs()[2]->cpu_data()[0]) << "already in cudnn defintion";
    caffe_gpu_scal(bn_layers_[i]->blobs()[0]->count(),
                   1 / bn_layers_[i]->blobs()[2]->cpu_data()[0],
                   bn_layers_[i]->blobs()[0]->mutable_gpu_data());
    caffe_gpu_scal(bn_layers_[i]->blobs()[1]->count(),
                   1 / bn_layers_[i]->blobs()[2]->cpu_data()[0],
                   bn_layers_[i]->blobs()[1]->mutable_gpu_data());
    bn_layers_[i]->blobs()[2]->mutable_cpu_data()[0] = -bn_layers_[i]->blobs()[2]->cpu_data()[0];
  }

  for (size_t i = 0; i < bottle_bn_layers_.size(); i++)
  {
    CHECK(0 < bottle_bn_layers_[i]->blobs()[2]->cpu_data()[0]) << "already in cudnn defintion";
    caffe_gpu_scal(bottle_bn_layers_[i]->blobs()[0]->count(),
                   1 / bottle_bn_layers_[i]->blobs()[2]->cpu_data()[0],
                   bottle_bn_layers_[i]->blobs()[0]->mutable_gpu_data());
    caffe_gpu_scal(bottle_bn_layers_[i]->blobs()[1]->count(),
                   1 / bottle_bn_layers_[i]->blobs()[2]->cpu_data()[0],
                   bottle_bn_layers_[i]->blobs()[1]->mutable_gpu_data());
    bottle_bn_layers_[i]->blobs()[2]->mutable_cpu_data()[0] = -bottle_bn_layers_[i]->blobs()[2]->cpu_data()[0];
  }

}
#endif // USE_CUDNN

template <typename Dtype>
void DenseBlockLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
  const vector<Blob<Dtype>*>& top)
{
  const vector<int> shape(bottom[0]->shape());
  const int n = shape[0];
  const int k0= shape[1];
  const int h = shape[2];
  const int w = shape[3];

  CHECK_EQ(k0 + num_layers_ * growth_rate_, top[0]->shape()[1])
    << "Invalid top shape according to k0 + num_layers_ * growth_rate_";

#ifdef USE_CUDNN
  if (this->phase_ == TEST)
  {
    this->ForwardInference_gpu(bottom, top);
    return;
  }
#endif // USE_CUDNN

  // copy the input data into working space 
  caffe_copy(bottom[0]->count(), bottom[0]->gpu_data(), maps_diff_.mutable_gpu_data());

#ifdef USE_CUDNN

  // we use BN's blobs[2] to mark whether the params is in caffe
  // definition all cudnn definition 
  // in caffe, blobs[2] > 0, mean = blobs[0] / blobs[2], var = blobs[1] / blobs[1]
  // in cudnn, blobs[2] < 0, mean = blobs[0], var = blobs[1]
  // cudnn=>caffe, blobs[2] = -blobs[2], blobs[0] *= blobs[2], blobs[1] *= blobs[2]
  // caffe=>cudnn, blobs[0] /= blobs[2], blobs[1] /= blobs[2], blobs[2] = -blobs[2]
  if (pre_bn_layer_->blobs()[2]->cpu_data()[0] > 0)
    convertBatchNormParams(); 

  Dtype one(1.f), zero(0.f);

  for (int l = 0; l < num_layers_; l++)
  {
    vector<Blob<Dtype>*> the_input_lth(1, input_lth_[l].get());
    vector<Blob<Dtype>*> the_conv3x3_inter_l(1, conv3x3_inter_[l].get());
    vector<Blob<Dtype>*> the_output_lth(1, output_lth_[l].get());

    if (use_bottleneck_)
    {
      vector<Blob<Dtype>*> the_bottleneck_inter_l(1, bottleneck_inter_[l].get());

      {
        updateMovingAverage( (BatchNormLayer<Dtype>*)(bottle_bn_layers_[l].get()) );
        CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
          cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero,
          input_desc_[l], the_input_lth[0]->gpu_data(),
          input_desc_[l], the_conv3x3_inter_l[0]->mutable_gpu_data(),
          input_scale_bias_desc_[l], bottle_scale_layers_[l]->blobs()[0]->gpu_data(), bottle_scale_layers_[l]->blobs()[1]->gpu_data(),
          1 / bottle_bn_layers_[l]->blobs()[2]->cpu_data()[0],
          bottle_bn_layers_[l]->blobs()[0]->mutable_gpu_data(),
          bottle_bn_layers_[l]->blobs()[1]->mutable_gpu_data(),
          cudnnGetBNEps(bottle_bn_layers_[l]->layer_param().batch_norm_param().eps()),
          bottleneck_bn_mean_var_[l]->mutable_gpu_data(), bottleneck_bn_mean_var_[l]->mutable_gpu_diff()
        ));
      }
      //bottle_bn_layers_[l]->Forward(the_input_lth, the_conv3x3_inter_l);
      // (in gpu) async "assemble" (original part) can start here to prepare for next conv block
      assemble_maps_gpu_origin_part(n, h, w, k0 + l * growth_rate_, growth_rate_,
        maps_diff_.mutable_gpu_data(), (const Dtype*)NULL /*output_lth_[l]->gpu_data()*/, dataCopyStream_);

      //bottle_scale_layers_[l]->Forward(the_conv3x3_inter_l, the_conv3x3_inter_l);
      bottle_relu_layers_[l]->Forward(the_conv3x3_inter_l, the_conv3x3_inter_l);

      conv1x1_layers_[l]->Forward(the_conv3x3_inter_l, the_bottleneck_inter_l);

      //bn_layers_[l]->Forward(the_bottleneck_inter_l, the_bottleneck_inter_l);
      //scale_layers_[l]->Forward(the_bottleneck_inter_l, the_bottleneck_inter_l);
      {
        caffe_copy(the_bottleneck_inter_l[0]->count(),
                   the_bottleneck_inter_l[0]->gpu_data(), 
                   bottleneck_scale_tmp_[l]->mutable_gpu_data());
        updateMovingAverage( (BatchNormLayer<Dtype>*)(bn_layers_[l].get()) );
        CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
          cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero,
          bottleneck_inter_desc_, bottleneck_scale_tmp_[l]->gpu_data(),
          bottleneck_inter_desc_, the_bottleneck_inter_l[0]->mutable_gpu_data(),
          bottleneck_scale_bias_desc_, scale_layers_[l]->blobs()[0]->gpu_data(), scale_layers_[l]->blobs()[1]->gpu_data(),
          1 / bn_layers_[l]->blobs()[2]->cpu_data()[0],
          bn_layers_[l]->blobs()[0]->mutable_gpu_data(),
          bn_layers_[l]->blobs()[1]->mutable_gpu_data(),
          cudnnGetBNEps(bn_layers_[l]->layer_param().batch_norm_param().eps()),
          bn_mean_var_[l]->mutable_gpu_data(), bn_mean_var_[l]->mutable_gpu_diff()
        ));
      }
      relu_layers_[l]->Forward(the_bottleneck_inter_l, the_bottleneck_inter_l);

      conv3x3_layers_[l]->Forward(the_bottleneck_inter_l, the_output_lth);

    }
    else
    {
      {
        updateMovingAverage( (BatchNormLayer<Dtype>*)(bn_layers_[l].get()) );
        CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
          cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero,
          input_desc_[l], the_input_lth[0]->gpu_data(),
          input_desc_[l], the_conv3x3_inter_l[0]->mutable_gpu_data(),
          input_scale_bias_desc_[l], scale_layers_[l]->blobs()[0]->gpu_data(), scale_layers_[l]->blobs()[1]->gpu_data(),
          1 / bn_layers_[l]->blobs()[2]->cpu_data()[0],
          bn_layers_[l]->blobs()[0]->mutable_gpu_data(),
          bn_layers_[l]->blobs()[1]->mutable_gpu_data(),
          cudnnGetBNEps(bn_layers_[l]->layer_param().batch_norm_param().eps()),
          bn_mean_var_[l]->mutable_gpu_data(), bn_mean_var_[l]->mutable_gpu_diff()
        ));
      }
      //bn_layers_[l]->Forward(the_input_lth, the_conv3x3_inter_l);

      // (in gpu) async "assemble" (original part) can start here to prepare for next conv block
      assemble_maps_gpu_origin_part(n, h, w, k0 + l * growth_rate_, growth_rate_,
        maps_diff_.mutable_gpu_data(), (const Dtype*)NULL /*output_lth_[l]->gpu_data()*/, dataCopyStream_);

      //scale_layers_[l]->Forward(the_conv3x3_inter_l, the_conv3x3_inter_l);
      relu_layers_[l]->Forward(the_conv3x3_inter_l, the_conv3x3_inter_l);

      conv3x3_layers_[l]->Forward(the_conv3x3_inter_l, the_output_lth);

    }

    if (use_dropout_)
    {
      dropout_layers_[l]->Forward(the_output_lth, the_output_lth);
    }


    // (in gpu) start async "assemble" (adding part) for this conv block
    //assemble_maps(n, h, w, k0 + l * growth_rate_, growth_rate_,
    //  maps_diff_.mutable_cpu_data(), output_lth_[l]->cpu_data());
    assemble_maps_gpu_adding_part(n, h, w, k0 + l * growth_rate_, growth_rate_,
      maps_diff_.mutable_gpu_data(), output_lth_[l]->gpu_data(), dataCopyStream_);

    // (in gpu) synchronize "assemble" here so we can start next conv block
    CUDA_CHECK(hipStreamSynchronize(dataCopyStream_));
  }

  // maps_diff_.data() store the output data (before pre_bn_layer_), 
  // which will be used in backward of the input scale of each conv block
  // Caffe's BatchNormLayer does not touch its top data in Backward, and we will use it to 
  // infer the input of 1st conv in every conv block

  //pre_bn_layer_->Forward(vector<Blob<Dtype>*>(1, &maps_diff_), vector<Blob<Dtype>*>(1, &maps_diff_));
  //post_scale_layer_->Forward(vector<Blob<Dtype>*>(1, &maps_diff_), top);
  {
    updateMovingAverage( (BatchNormLayer<Dtype>*)(pre_bn_layer_.get()) );
    CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
      cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero,
      final_output_desc_, maps_diff_.gpu_data(),
      final_output_desc_, top[0]->mutable_gpu_data(),
      scale_bias_desc_, post_scale_layer_->blobs()[0]->gpu_data(), post_scale_layer_->blobs()[1]->gpu_data(),
      1 / pre_bn_layer_->blobs()[2]->cpu_data()[0],
      pre_bn_layer_->blobs()[0]->mutable_gpu_data(),
      pre_bn_layer_->blobs()[1]->mutable_gpu_data(),
      cudnnGetBNEps(pre_bn_layer_->layer_param().batch_norm_param().eps()),
      pre_bn_mean_var_.mutable_gpu_data(), pre_bn_mean_var_.mutable_gpu_diff()
    ));
  }
  post_relu_layer_->Forward(top, top);

#else // USE_CUDNN

  for (int l = 0; l < num_layers_; l++)
  {
    vector<Blob<Dtype>*> the_input_lth(1, input_lth_[l].get());
    vector<Blob<Dtype>*> the_conv3x3_inter_l(1, conv3x3_inter_[l].get());
    vector<Blob<Dtype>*> the_output_lth(1, output_lth_[l].get());

    if (use_bottleneck_)
    {
      vector<Blob<Dtype>*> the_bottleneck_inter_l(1, bottleneck_inter_[l].get());

	  bottle_bn_layers_[l]->Forward(the_input_lth, the_conv3x3_inter_l);
	  // (in gpu) async "assemble" (original part) can start here to prepare for next conv block
      assemble_maps_gpu_origin_part(n, h, w, k0 + l * growth_rate_, growth_rate_,
        maps_diff_.mutable_gpu_data(), (const Dtype*)NULL /*output_lth_[l]->gpu_data()*/, dataCopyStream_);

	  bottle_scale_layers_[l]->Forward(the_conv3x3_inter_l, the_conv3x3_inter_l);
	  bottle_relu_layers_[l]->Forward(the_conv3x3_inter_l, the_conv3x3_inter_l);

      conv1x1_layers_[l]->Forward(the_conv3x3_inter_l, the_bottleneck_inter_l);

	  bn_layers_[l]->Forward(the_bottleneck_inter_l, the_bottleneck_inter_l);
	  scale_layers_[l]->Forward(the_bottleneck_inter_l, the_bottleneck_inter_l);
      relu_layers_[l]->Forward(the_bottleneck_inter_l, the_bottleneck_inter_l);

      conv3x3_layers_[l]->Forward(the_bottleneck_inter_l, the_output_lth);

    }
    else
    {
      bn_layers_[l]->Forward(the_input_lth, the_conv3x3_inter_l);

      // (in gpu) async "assemble" (original part) can start here to prepare for next conv block
      assemble_maps_gpu_origin_part(n, h, w, k0 + l * growth_rate_, growth_rate_,
        maps_diff_.mutable_gpu_data(), (const Dtype*)NULL /*output_lth_[l]->gpu_data()*/, dataCopyStream_);

	  scale_layers_[l]->Forward(the_conv3x3_inter_l, the_conv3x3_inter_l);
	  relu_layers_[l]->Forward(the_conv3x3_inter_l, the_conv3x3_inter_l);

      conv3x3_layers_[l]->Forward(the_conv3x3_inter_l, the_output_lth);

    }

    if (use_dropout_)
    {
      dropout_layers_[l]->Forward(the_output_lth, the_output_lth);
    }


    // (in gpu) start async "assemble" (adding part) for this conv block
    //assemble_maps(n, h, w, k0 + l * growth_rate_, growth_rate_,
    //  maps_diff_.mutable_cpu_data(), output_lth_[l]->cpu_data());
    assemble_maps_gpu_adding_part(n, h, w, k0 + l * growth_rate_, growth_rate_,
      maps_diff_.mutable_gpu_data(), output_lth_[l]->gpu_data(), dataCopyStream_);

    // (in gpu) synchronize "assemble" here so we can start next conv block
    CUDA_CHECK(hipStreamSynchronize(dataCopyStream_));
  }

  // maps_diff_.data() store the output data (before post_scale_layer_, after pre_bn_layer_), 
  // which will be used in backward of the input scale of each conv block
  // Caffe's BatchNormLayer does not touch its top data in Backward, and we will use it to 
  // infer the input of 1st conv in every conv block

  pre_bn_layer_->Forward(vector<Blob<Dtype>*>(1, &maps_diff_), vector<Blob<Dtype>*>(1, &maps_diff_));
  post_scale_layer_->Forward(vector<Blob<Dtype>*>(1, &maps_diff_), top);
  post_relu_layer_->Forward(top, top);

#endif // USE_CUDNN

}

template <typename Dtype>
void DenseBlockLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
{
  const vector<int> shape(top[0]->shape());
  const int n = shape[0];
  const int k0= shape[1] - num_layers_ * growth_rate_;
  const int h = shape[2];
  const int w = shape[3];
  
  CHECK_EQ(k0, bottom[0]->shape()[1])
    << "Invalid top shape according to k0 + num_layers_ * growth_rate_";

#ifdef USE_CUDNN

  Dtype one(1.f), zero(0.f);

  post_relu_layer_->Backward(top, need_propagate_down_, top);
  //post_scale_layer_->Backward(top, need_propagate_down_, vector<Blob<Dtype>*>(1, &maps_diff_));
  //pre_bn_layer_->Backward(vector<Blob<Dtype>*>(1, &maps_diff_), need_propagate_down_, vector<Blob<Dtype>*>(1, &maps_diff_));
  // maps_diff still hold the data of feature maps (before BN)
  {
    CUDNN_CHECK(hipdnnBatchNormalizationBackward(
      cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero, &one, &one, 
      final_output_desc_, maps_diff_.gpu_data(), 
      final_output_desc_, top[0]->gpu_diff(), 
      final_output_desc_, maps_diff_.mutable_gpu_diff(), 
      scale_bias_desc_, post_scale_layer_->blobs()[0]->gpu_data(),
      post_scale_layer_->blobs()[0]->mutable_gpu_diff(),
      post_scale_layer_->blobs()[1]->mutable_gpu_diff(),
      cudnnGetBNEps(pre_bn_layer_->layer_param().batch_norm_param().eps()),
      pre_bn_mean_var_.gpu_data(), pre_bn_mean_var_.gpu_diff()
    ));
  }

  for (int l = num_layers_ - 1; l >= 0; l--)
  {
    vector<Blob<Dtype>*> the_input_lth(1, input_lth_[l].get());
    vector<Blob<Dtype>*> the_conv3x3_inter_l(1, conv3x3_inter_[l].get());
    vector<Blob<Dtype>*> the_output_lth(1, output_lth_[l].get());

    // diff and data, each use a individual stream
    // (in gpu) start async "disassemble" (adding part) for this conv block
    disassemble_maps_gpu_adding_part(n, h, w, k0 + l * growth_rate_, growth_rate_,
      maps_diff_.mutable_gpu_data(), output_lth_[l]->mutable_gpu_data(), dataCopyStream_);
    disassemble_maps_gpu_adding_part(n, h, w, k0 + l * growth_rate_, growth_rate_,
      maps_diff_.mutable_gpu_diff(), output_lth_[l]->mutable_gpu_diff(), diffCopyStream_);

    // (in gpu) synchronize "disassemble" (adding part) here so we can start the 
    // Backward for the conv block
    CUDA_CHECK(hipStreamSynchronize(dataCopyStream_));
    CUDA_CHECK(hipStreamSynchronize(diffCopyStream_));

    // (in gpu) start async "disassemble" (original part) to prepare for Backward of 
    // earlier conv in the conv block
    disassemble_maps_gpu_origin_part(n, h, w, k0 + l * growth_rate_, growth_rate_,
      maps_diff_.mutable_gpu_data(), (Dtype*)NULL /*output_lth_[l]->mutable_gpu_data()*/, dataCopyStream_);
    disassemble_maps_gpu_origin_part(n, h, w, k0 + l * growth_rate_, growth_rate_,
      maps_diff_.mutable_gpu_diff(), (Dtype*)NULL /*output_lth_[l]->mutable_gpu_diff()*/, diffCopyStream_);


    if (use_dropout_)
    {
      dropout_layers_[l]->Backward(the_output_lth, need_propagate_down_, the_output_lth);
    }

    if (use_bottleneck_)
    {
      vector<Blob<Dtype>*> the_bottleneck_inter_l(1, bottleneck_inter_[l].get());

      conv3x3_layers_[l]->Backward(the_output_lth, need_propagate_down_, the_bottleneck_inter_l);

      relu_layers_[l]->Backward(the_bottleneck_inter_l, need_propagate_down_, the_bottleneck_inter_l);
      //scale_layers_[l]->Backward(the_bottleneck_inter_l, need_propagate_down_, the_bottleneck_inter_l);
      //bn_layers_[l]->Backward(the_bottleneck_inter_l, need_propagate_down_, the_bottleneck_inter_l);
      {
        CUDNN_CHECK(hipdnnBatchNormalizationBackward(
          cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero, &one, &one, 
          bottleneck_inter_desc_, bottleneck_scale_tmp_[l]->gpu_data(), 
          bottleneck_inter_desc_, the_bottleneck_inter_l[0]->gpu_diff(), 
          bottleneck_inter_desc_, bottleneck_scale_tmp_[l]->mutable_gpu_diff(), 
          bottleneck_scale_bias_desc_, scale_layers_[l]->blobs()[0]->gpu_data(),
          scale_layers_[l]->blobs()[0]->mutable_gpu_diff(),
          scale_layers_[l]->blobs()[1]->mutable_gpu_diff(),
          cudnnGetBNEps(bn_layers_[l]->layer_param().batch_norm_param().eps()),
          bn_mean_var_[l]->gpu_data(), bn_mean_var_[l]->gpu_diff()
        ));
        caffe_copy(the_bottleneck_inter_l[0]->count(),
                   bottleneck_scale_tmp_[l]->gpu_diff(),
                   the_bottleneck_inter_l[0]->mutable_gpu_diff());
      }

      // (in gpu) synchronize "disassemble" (original part) so we can continue the preparation 
      // for Backward of conv3x3 in the conv block
      CUDA_CHECK(hipStreamSynchronize(dataCopyStream_));
      CUDA_CHECK(hipStreamSynchronize(diffCopyStream_));

      // re-calculate the bottom_data of conv1x1 from bottle_scale_layers_[l] and bottle_relu_layers_[l]
      // input_lth_[l] (maps_diff_) hold the data after BN, only scale is needed
      //bottle_scale_layers_[l]->Forward(the_input_lth, the_conv3x3_inter_l);
      {
        Dtype *tmp_space = pre_bn_mean_var_.mutable_gpu_diff();
        caffe_gpu_powx(
          bottleneck_bn_mean_var_[l]->count(), 
          bottleneck_bn_mean_var_[l]->gpu_diff(), 
          Dtype(-2.), tmp_space
        ); // Backward of pre_bn_layer_ is complete, space of pre_bn_mean_var_ can be reuse
        CUDNN_CHECK(hipdnnBatchNormalizationForwardInference(
          cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero,
          input_desc_[l], the_input_lth[0]->gpu_data(),
          input_desc_[l], the_conv3x3_inter_l[0]->mutable_gpu_data(),
          input_scale_bias_desc_[l], bottle_scale_layers_[l]->blobs()[0]->gpu_data(), bottle_scale_layers_[l]->blobs()[1]->gpu_data(),
          bottleneck_bn_mean_var_[l]->gpu_data(), tmp_space,
          cudnnGetBNEps(bottle_bn_layers_[l]->layer_param().batch_norm_param().eps())
        ));
      }
      bottle_relu_layers_[l]->Forward(the_conv3x3_inter_l, the_conv3x3_inter_l);

      conv1x1_layers_[l]->Backward(the_bottleneck_inter_l, need_propagate_down_, the_conv3x3_inter_l);

      bottle_relu_layers_[l]->Backward(the_conv3x3_inter_l, need_propagate_down_, the_conv3x3_inter_l);
      //bottle_scale_layers_[l]->Backward(the_conv3x3_inter_l, need_propagate_down_, the_conv3x3_inter_l);
      //bottle_bn_layers_[l]->Backward(the_conv3x3_inter_l, need_propagate_down_, the_input_lth);
      {
        CUDNN_CHECK(hipdnnBatchNormalizationBackward(
          cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero, &one, &one, 
          input_desc_[l], the_input_lth[0]->gpu_data(), 
          input_desc_[l], the_conv3x3_inter_l[0]->gpu_diff(), 
          input_desc_[l], the_input_lth[0]->mutable_gpu_diff(), 
          input_scale_bias_desc_[l], bottle_scale_layers_[l]->blobs()[0]->gpu_data(),
          bottle_scale_layers_[l]->blobs()[0]->mutable_gpu_diff(),
          bottle_scale_layers_[l]->blobs()[1]->mutable_gpu_diff(),
          cudnnGetBNEps(bottle_bn_layers_[l]->layer_param().batch_norm_param().eps()),
          bottleneck_bn_mean_var_[l]->gpu_data(), bottleneck_bn_mean_var_[l]->gpu_diff()
        ));
      }
    }
    else
    {
      // (in gpu) synchronize "disassemble" (original part) so we can continue the preparation 
      // for Backward of conv3x3 in the conv block
      CUDA_CHECK(hipStreamSynchronize(dataCopyStream_));
      CUDA_CHECK(hipStreamSynchronize(diffCopyStream_));

      // re-calculate the bottom_data of conv3x3 from scale_layers_[l] and relu_layers_[l]

      //scale_layers_[l]->Forward(the_input_lth, the_conv3x3_inter_l);
      {
        Dtype *tmp_space = pre_bn_mean_var_.mutable_gpu_diff();
        caffe_gpu_powx(
          bn_mean_var_[l]->count(), 
          bn_mean_var_[l]->gpu_diff(), 
          Dtype(-2.), tmp_space
        ); // Backward of pre_bn_layer_ is complete, space of pre_bn_mean_var_ can be reuse
        CUDNN_CHECK(hipdnnBatchNormalizationForwardInference(
          cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero,
          input_desc_[l], the_input_lth[0]->gpu_data(),
          input_desc_[l], the_conv3x3_inter_l[0]->mutable_gpu_data(),
          input_scale_bias_desc_[l], scale_layers_[l]->blobs()[0]->gpu_data(), scale_layers_[l]->blobs()[1]->gpu_data(),
          bn_mean_var_[l]->gpu_data(), tmp_space,
          cudnnGetBNEps(bn_layers_[l]->layer_param().batch_norm_param().eps())
        ));
      }
      relu_layers_[l]->Forward(the_conv3x3_inter_l, the_conv3x3_inter_l);

      conv3x3_layers_[l]->Backward(the_output_lth, need_propagate_down_, the_conv3x3_inter_l);

      relu_layers_[l]->Backward(the_conv3x3_inter_l, need_propagate_down_, the_conv3x3_inter_l);
      //scale_layers_[l]->Backward(the_conv3x3_inter_l, need_propagate_down_, the_conv3x3_inter_l);
      //bn_layers_[l]->Backward(the_conv3x3_inter_l, need_propagate_down_, the_input_lth);
      {
        CUDNN_CHECK(hipdnnBatchNormalizationBackward(
          cudnn_handle_, HIPDNN_BATCHNORM_SPATIAL, &one, &zero, &one, &one, 
          input_desc_[l], the_input_lth[0]->gpu_data(), 
          input_desc_[l], the_conv3x3_inter_l[0]->gpu_diff(), 
          input_desc_[l], the_input_lth[0]->mutable_gpu_diff(), 
          input_scale_bias_desc_[l], scale_layers_[l]->blobs()[0]->gpu_data(),
          scale_layers_[l]->blobs()[0]->mutable_gpu_diff(),
          scale_layers_[l]->blobs()[1]->mutable_gpu_diff(),
          cudnnGetBNEps(bn_layers_[l]->layer_param().batch_norm_param().eps()),
          bn_mean_var_[l]->gpu_data(), bn_mean_var_[l]->gpu_diff()
        ));
      }
    }

    { // add the diff together before continue
      const int count = input_lth_[l]->count();
      Dtype* target_ptr;
      const Dtype* adding_in_ptr;
      //if (l > 0) // in the original structure we will always copy the diff to bottom.diff
      {
        target_ptr = maps_diff_.mutable_gpu_diff();
        adding_in_ptr = tmp_diff_.gpu_diff(); // diff of input_lth_[l]
      }
      //else
      //{
      //  // for the first conv block, store the sum of diff in tmp_diff_.diff (input_lth_[0].diff)
      //  // because pre_bn_layer_ treat input_lth_[0] as the top blob.
      //  target_ptr = tmp_diff_.mutable_gpu_diff(); // diff of input_lth_[l]
      //  adding_in_ptr = maps_diff_.gpu_diff();
      //}

      // in gpu caffe_gpu_axpy is used
      //caffe_axpy(count, Dtype(1.), adding_in_ptr, target_ptr);
      caffe_gpu_axpy(count, Dtype(1.), adding_in_ptr, target_ptr);
    }
  }

#else // USE_CUDNN

  post_relu_layer_->Backward(top, need_propagate_down_, top);
  post_scale_layer_->Backward(top, need_propagate_down_, vector<Blob<Dtype>*>(1, &maps_diff_));
  pre_bn_layer_->Backward(vector<Blob<Dtype>*>(1, &maps_diff_), need_propagate_down_, vector<Blob<Dtype>*>(1, &maps_diff_));
  // maps_diff still hold the data of feature maps (after BN, before Scale)

  for (int l = num_layers_ - 1; l >= 0 ; l --)
  {
    vector<Blob<Dtype>*> the_input_lth(1, input_lth_[l].get());
    vector<Blob<Dtype>*> the_conv3x3_inter_l(1, conv3x3_inter_[l].get());
    vector<Blob<Dtype>*> the_output_lth(1, output_lth_[l].get());
    
    // diff and data, each use a individual stream
    // (in gpu) start async "disassemble" (adding part) for this conv block
    disassemble_maps_gpu_adding_part(n, h, w, k0 + l * growth_rate_, growth_rate_, 
                     maps_diff_.mutable_gpu_data(), output_lth_[l]->mutable_gpu_data(), dataCopyStream_);
    disassemble_maps_gpu_adding_part(n, h, w, k0 + l * growth_rate_, growth_rate_, 
                     maps_diff_.mutable_gpu_diff(), output_lth_[l]->mutable_gpu_diff(), diffCopyStream_);
    
    // (in gpu) synchronize "disassemble" (adding part) here so we can start the 
    // Backward for the conv block
    CUDA_CHECK(hipStreamSynchronize(dataCopyStream_));
    CUDA_CHECK(hipStreamSynchronize(diffCopyStream_));

    // (in gpu) start async "disassemble" (original part) to prepare for Backward of 
    // earlier conv in the conv block
    disassemble_maps_gpu_origin_part(n, h, w, k0 + l * growth_rate_, growth_rate_,
      maps_diff_.mutable_gpu_data(), (Dtype*)NULL /*output_lth_[l]->mutable_gpu_data()*/, dataCopyStream_);
    disassemble_maps_gpu_origin_part(n, h, w, k0 + l * growth_rate_, growth_rate_,
      maps_diff_.mutable_gpu_diff(), (Dtype*)NULL /*output_lth_[l]->mutable_gpu_diff()*/, diffCopyStream_);
    
    
    if (use_dropout_)
    {
      dropout_layers_[l]->Backward(the_output_lth, need_propagate_down_, the_output_lth);
    }
    
    if (use_bottleneck_)
    {
      vector<Blob<Dtype>*> the_bottleneck_inter_l(1, bottleneck_inter_[l].get());
      
      conv3x3_layers_[l]->Backward(the_output_lth, need_propagate_down_, the_bottleneck_inter_l);
      
      relu_layers_[l]->Backward(the_bottleneck_inter_l, need_propagate_down_, the_bottleneck_inter_l);
      scale_layers_[l]->Backward(the_bottleneck_inter_l, need_propagate_down_, the_bottleneck_inter_l);
      bn_layers_[l]->Backward(the_bottleneck_inter_l, need_propagate_down_, the_bottleneck_inter_l);

      // (in gpu) synchronize "disassemble" (original part) so we can continue the preparation 
      // for Backward of conv3x3 in the conv block
      CUDA_CHECK(hipStreamSynchronize(dataCopyStream_));
      CUDA_CHECK(hipStreamSynchronize(diffCopyStream_));
      
      // re-calculate the bottom_data of conv1x1 from bottle_scale_layers_[l] and bottle_relu_layers_[l]
      // input_lth_[l] (maps_diff_) hold the data after BN, only scale is needed
      bottle_scale_layers_[l]->Forward(the_input_lth, the_conv3x3_inter_l);
      bottle_relu_layers_[l]->Forward(the_conv3x3_inter_l, the_conv3x3_inter_l);
      
      conv1x1_layers_[l]->Backward(the_bottleneck_inter_l, need_propagate_down_, the_conv3x3_inter_l);
      
      bottle_relu_layers_[l]->Backward(the_conv3x3_inter_l, need_propagate_down_, the_conv3x3_inter_l);
      bottle_scale_layers_[l]->Backward(the_conv3x3_inter_l, need_propagate_down_, the_conv3x3_inter_l);
      bottle_bn_layers_[l]->Backward(the_conv3x3_inter_l, need_propagate_down_, the_input_lth);
    }
    else
    {
      // (in gpu) synchronize "disassemble" (original part) so we can continue the preparation 
      // for Backward of conv3x3 in the conv block
      CUDA_CHECK(hipStreamSynchronize(dataCopyStream_));
      CUDA_CHECK(hipStreamSynchronize(diffCopyStream_));
      
      // re-calculate the bottom_data of conv3x3 from scale_layers_[l] and relu_layers_[l]

      scale_layers_[l]->Forward(the_input_lth, the_conv3x3_inter_l);
      relu_layers_[l]->Forward(the_conv3x3_inter_l, the_conv3x3_inter_l);
      
      conv3x3_layers_[l]->Backward(the_output_lth, need_propagate_down_, the_conv3x3_inter_l);
      
      relu_layers_[l]->Backward(the_conv3x3_inter_l, need_propagate_down_, the_conv3x3_inter_l);
      scale_layers_[l]->Backward(the_conv3x3_inter_l, need_propagate_down_, the_conv3x3_inter_l);
      bn_layers_[l]->Backward(the_conv3x3_inter_l, need_propagate_down_, the_input_lth);
    }
    
    { // add the diff together before continue
      const int count = input_lth_[l]->count();
      Dtype* target_ptr;
      const Dtype* adding_in_ptr;
      //if (l > 0) // in the original structure we will always copy the diff to bottom.diff
      {
        target_ptr = maps_diff_.mutable_gpu_diff(); 
        adding_in_ptr = tmp_diff_.gpu_diff(); // diff of input_lth_[l]
      }
      //else
      //{
      //  // for the first conv block, store the sum of diff in tmp_diff_.diff (input_lth_[0].diff)
      //  // because pre_bn_layer_ treat input_lth_[0] as the top blob.
      //  target_ptr = tmp_diff_.mutable_gpu_diff(); // diff of input_lth_[l]
      //  adding_in_ptr = maps_diff_.gpu_diff();
      //}
      
      // in gpu caffe_gpu_axpy is used
      //caffe_axpy(count, Dtype(1.), adding_in_ptr, target_ptr);
      caffe_gpu_axpy(count, Dtype(1.), adding_in_ptr, target_ptr);
    }
  }

#endif // USE_CUDNN
  
  caffe_copy(bottom[0]->count(), maps_diff_.gpu_diff(), bottom[0]->mutable_gpu_diff());
}

INSTANTIATE_LAYER_GPU_FUNCS(DenseBlockLayer);

} // namespace caffe
